#include "common.h"

namespace hagrid {

__host__ float profile(std::function<void()> f) {
    hipEvent_t start_kernel, end_kernel;
    CHECK_CUDA_CALL(hipEventCreate(&start_kernel));
    CHECK_CUDA_CALL(hipEventCreate(&end_kernel));
    CHECK_CUDA_CALL(hipEventRecord(start_kernel));
    f();
    CHECK_CUDA_CALL(hipEventRecord(end_kernel));
    CHECK_CUDA_CALL(hipEventSynchronize(end_kernel));
    float kernel_time = 0;
    CHECK_CUDA_CALL(hipEventElapsedTime(&kernel_time, start_kernel, end_kernel));
    CHECK_CUDA_CALL(hipEventDestroy(start_kernel));
    CHECK_CUDA_CALL(hipEventDestroy(end_kernel));
    return kernel_time;
}

} // namespace hagrid
